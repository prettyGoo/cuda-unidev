#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <stdio.h>
#include <iostream>

using namespace std;

hipError_t addWithCuda(float *c, float *a, float *b, unsigned int size);

__global__ void addKernel(float *c, float *a, float *b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	srand(time(0));

	unsigned int start_time = clock(); // ��������� �����

	int arraySize = 1000000;
	float *a = new float[arraySize];
	float *b = new float[arraySize];
	float *c = new float[arraySize];
	for (int i = 0; i < arraySize; i++) {
		a[i] = rand() % 10;
		b[i] = rand() % 10;
		c[i] = 0;
	}
	
	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("Array size = %d\n", arraySize);


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	unsigned int end_time = clock(); // �������� �����

	cout << "Time: " << end_time - start_time << endl;
	system("pause");
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *c, float *a, float *b, unsigned int size)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <10000, 10 >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
