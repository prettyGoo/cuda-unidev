#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <ctime>
#include <iostream>

using namespace std;

hipError_t mulWithCuda(float *A, float *B, float *C, int size);

__global__ void matrixMul(float *A, float *B, float *C, int size)
{
	unsigned __int16 i = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned __int16 j = threadIdx.y + blockIdx.y*blockDim.y;
	float sum = 0.0;

	for (unsigned __int16 k = 0; k < size; k++) {
		sum += A[k + i*size] * B[j + k*size];
	}

	C[j + i*size] = sum;
}

void initArray(float *arr, int size) {
	for (int i = 0; i < size; i++)
		arr[i] = rand() % 10;
}

int main()
{
	srand(time(0));
	const int arraySize = 4;
	float *A = new float[arraySize*arraySize];
	float *B = new float[arraySize*arraySize];
	float *C = new float[arraySize*arraySize];

	initArray(A, arraySize*arraySize);
	initArray(B, arraySize*arraySize);

	// Add vectors in parallel.
	hipError_t cudaStatus = mulWithCuda(A, B, C, arraySize);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();

	for (int i = 0; i < arraySize*arraySize; i++) {
		cout << C[i];
	}

	delete A;
	delete B;
	delete C;

	system("pause");

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t mulWithCuda(float *A, float *B, float *C, int size)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, A, size * sizeof(int), hipMemcpyHostToDevice);

	cudaStatus = hipMemcpy(dev_b, B, size * sizeof(int), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU with one thread for each element.
	matrixMul << <1, size >> > (dev_a, dev_b, dev_c, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(C, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);


	return cudaStatus;
}
