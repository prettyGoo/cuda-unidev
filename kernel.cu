#include "hip/hip_runtime.h"


#include <ctime>
#include <stdio.h>
#include <iostream>

using namespace std;

const unsigned int arrsize = 160;
const int BLOCK_SIZE = 16;

void initArr(double* arr, bool fill_with_zero);
void multiplyMatrixes(double* firstArr, double* secondArr, double* finalArr);
double sumArrayElems(double *arr);


//=================== GPU ===================
__global__ void matrixmul_kernel(double *A, double *B, double *C) {
	double sum = 0;

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int k = 0; k < arrsize; k++) {
		sum += A[row * arrsize + k] * B[k * arrsize + col];
	}

	C[row * arrsize + col] = sum;
}


int main()
{
	srand(time(0));

	double *A = new double[arrsize*arrsize];
	double *B = new double[arrsize*arrsize];
	double *C = new double[arrsize*arrsize];

	initArr(A, false);
	initArr(B, false);
	initArr(C, true);

	printf("The size of elems in A and B is %d x %d", arrsize, arrsize);
	//=================== CPU //===================
	cout << endl << "CPU" << endl;

	clock_t start_time = clock();
	multiplyMatrixes(A, B, C);
	clock_t end_time = clock();

	cout << "Time: " << end_time - start_time << endl;
	cout << "Sum of the elements after CPU concat: " << sumArrayElems(C) << endl;

	//=================== GPU ===================
	cout << endl << "GPU" << endl;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	size_t raw_size = arrsize * arrsize * sizeof(double);

	double *aDevice = nullptr;
	double *bDevice = nullptr;
	double *cDevice = nullptr;

	// Выделить память под 
	hipMalloc((void**)&aDevice, raw_size);
	hipMalloc((void**)&bDevice, raw_size);
	hipMalloc((void**)&cDevice, raw_size);

	// Копировать массивы A и B в память GPU
	hipMemcpy(aDevice, A, raw_size, hipMemcpyHostToDevice);
	hipMemcpy(bDevice, B, raw_size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((arrsize + dimBlock.x - 1) / dimBlock.x, (arrsize + dimBlock.y - 1) / dimBlock.y);

	hipEventRecord(start);
	matrixmul_kernel <<<dimGrid, dimBlock>>> (aDevice, bDevice, cDevice);
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	hipDeviceSynchronize();
	hipMemcpy(C, cDevice, raw_size, hipMemcpyDeviceToHost);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "Time: " << milliseconds << endl;
	cout << "Sum of the elements after GPU concat: " << sumArrayElems(C) << endl;

	// ==========================================
	hipFree(aDevice);
	hipFree(bDevice);
	hipFree(cDevice);

	delete A;
	delete B;
	delete C;

	system("pause");
	return 0;
}

//=================== CPU ===================
// Функция, которая принимает на вход массив и инициализирует его рандомными значениями
void initArr(double* arr, bool fill_with_zero) {
	if (fill_with_zero) {
		for (int i = 0; i < arrsize*arrsize; i++)
			arr[i] = 0;
	}
	else {
		for (int i = 0; i < arrsize*arrsize; i++)
			arr[i] = 2;// rand() % 10;
	}

}

// Функция, которая складывает поэлементно два массива
void multiplyMatrixes(double* firstArr, double* secondArr, double* finalArr) {

	double row_sum = 0;
	double column_sum = 0;
	double sum = 0;

	int final_array_index = 0;

	// обходим каждую строку первой матрицы
	for (int i = 0; i < arrsize; i++) {
		// обходим каждый столбец второй матрицы
		for (int j = 0; j < arrsize; j++) {
			for (int k = 0; k < arrsize; k++) {
				sum += firstArr[i*k] * secondArr[j*k];
			}

			finalArr[final_array_index] = sum;
			sum = 0;
			final_array_index++;
		}
	}
}

// Функция, которая находит сумму всех элементов в массиве
double sumArrayElems(double* arr)
{
	double sum = 0;

	for (int i = 0; i < arrsize*arrsize; i++)
		sum += arr[i];
	return sum;
}
