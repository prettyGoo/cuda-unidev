#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <stdio.h>
#include <iostream>

using namespace std;

const unsigned int arrsize = 144;
const int BLOCK_SIZE = 16;

void initArr(double* arr, bool fill_with_zero);
void multiplyMatrixes(double* firstArr, double* secondArr, double* finalArr);
double sumArrayElems(double *arr);


//=================== GPU ===================
__global__ void matrixmul_kernel(double *A, double *B, double *C) {
	double sum = 0;

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int k = 0; k < arrsize; k++) {
		sum += A[row * arrsize + k] * B[k * arrsize + col];
	}

	C[row * arrsize + col] = sum;
}

//=================SHARED MEMORY=============
__global__ void matrixMul_shm(double *A, double *B, double *C, size_t size)
{
	//Вычисляемый элемент матрицы C
	double sum = 0.0;
	//Индекс начала первой и последней подматриц A, обрабатываемой блоком
	unsigned __int16 aBegin = size*BLOCK_SIZE*blockIdx.y;
	unsigned __int16 aEnd = aBegin + size - 1;
	//Шаг для вычисления индекса начала подматриц A
	unsigned __int16 aStep = BLOCK_SIZE;
	//Индекс начала первой подматрицы B, обрабатываемой блоком
	unsigned __int16 bBegin = BLOCK_SIZE*blockIdx.x;
	//Шаг для вычисления индекса начала подматриц B
	unsigned __int16 bStep = BLOCK_SIZE*size;
	//Цикл по 16*16 матрицам
	for (unsigned __int16 ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep)
	{
		//Очередная подматрица A в разделяемой памяти. Для размерности 16 каждая матрица будет занимать 1 Кб памяти.
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		//Очередная подматрица B в разделяесмой памяти.
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
		//Загрузить по одному элементу из A и B в разделяемую паямть
		As[threadIdx.y][threadIdx.x] = A[ia + size*threadIdx.y + threadIdx.x];
		Bs[threadIdx.y][threadIdx.x] = B[ib + size*threadIdx.y + threadIdx.x];
		//Дождаться, когда обе подматрицы будут полностью загружены
		__syncthreads();
		//Вычисляем нужный элемент произведения загруженных подматриц
		for (unsigned __int16 k = 0; k < BLOCK_SIZE; k++)
		{
			sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
		}
		//Дождаться, пока все остальные нити блока закончат вычислять свои элементы
		__syncthreads();
	}
	//Расчитать смещение индекса для матртцы C
	unsigned __int16 ic = size*BLOCK_SIZE*blockIdx.y + BLOCK_SIZE*blockIdx.x;
	//Записать результат в глобальную память
	C[ic + size*threadIdx.y + threadIdx.x] = sum;
}


int main()
{
	srand(time(0));

	double *A = new double[arrsize*arrsize];
	double *B = new double[arrsize*arrsize];
	double *C = new double[arrsize*arrsize];

	initArr(A, false);
	initArr(B, false);
	initArr(C, true);

	printf("The size of elems in A and B is %d x %d", arrsize, arrsize);
	//=================== CPU //===================
	cout << endl << "CPU" << endl;

	clock_t start_time = clock();
	multiplyMatrixes(A, B, C);
	clock_t end_time = clock();

	cout << "Time: " << end_time - start_time << endl;
	cout << "Sum of the elements after CPU concat: " << sumArrayElems(C) << endl;

	//=================== GPU ===================
	cout << endl << "GPU" << endl;

	initArr(C, true);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	size_t raw_size = arrsize * arrsize * sizeof(double);

	double *aDevice = nullptr;
	double *bDevice = nullptr;
	double *cDevice = nullptr;

	// Выделить память под 
	hipMalloc((void**)&aDevice, raw_size);
	hipMalloc((void**)&bDevice, raw_size);
	hipMalloc((void**)&cDevice, raw_size);

	// Копировать массивы A и B в память GPU
	hipMemcpy(aDevice, A, raw_size, hipMemcpyHostToDevice);
	hipMemcpy(bDevice, B, raw_size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(9, 9);

	hipEventRecord(start);
	matrixmul_kernel <<<dimGrid, dimBlock>>> (aDevice, bDevice, cDevice);
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	hipDeviceSynchronize();
	hipMemcpy(C, cDevice, raw_size, hipMemcpyDeviceToHost);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "Time: " << milliseconds << endl;
	cout << "Sum of the elements after GPU concat: " << sumArrayElems(C) << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);


	//=================SHARED MEMORY=============
	cout << endl << "SHARED MEMORY" << endl;

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(matrixMul_shm), hipFuncCachePreferShared);

	initArr(C, true);

	double *aDeviceS = nullptr;
	double *bDeviceS = nullptr;
	double *cDeviceS = nullptr;

	// Выделить память под 
	hipMalloc((void**)&aDeviceS, raw_size);
	hipMalloc((void**)&bDeviceS, raw_size);
	hipMalloc((void**)&cDeviceS, raw_size);

	// Копировать массивы A и B в память GPU
	hipMemcpy(aDeviceS, A, raw_size, hipMemcpyHostToDevice);
	hipMemcpy(bDeviceS, B, raw_size, hipMemcpyHostToDevice);

	hipEvent_t startS, stopS;
	hipEventCreate(&startS);
	hipEventCreate(&stopS);

	hipEventRecord(startS);
	//start_time = clock();
	matrixMul_shm << <dimGrid, dimBlock >> > (aDeviceS, bDeviceS, cDeviceS, arrsize);
	hipEventRecord(stopS);

	hipEventSynchronize(stopS);

	hipDeviceSynchronize();
	hipMemcpy(C, cDeviceS, raw_size, hipMemcpyDeviceToHost);

	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, startS, stopS);

	cout << "Time: " << milliseconds << endl;
	cout << "Sum of the elements after GPU concat (SHARED MEMORY): " << sumArrayElems(C) << endl;

	// ==========================================
	hipFree(aDevice);
	hipFree(bDevice);
	hipFree(cDevice);

	delete A;
	delete B;
	delete C;

	system("pause");
	return 0;
}

//=================== CPU ===================
// Функция, которая принимает на вход массив и инициализирует его рандомными значениями
void initArr(double* arr, bool fill_with_zero) {
	if (fill_with_zero) {
		for (int i = 0; i < arrsize*arrsize; i++)
			arr[i] = 0;
	}
	else {
		for (int i = 0; i < arrsize*arrsize; i++)
			//arr[i] = 2;
			arr[i] = rand() % 10;
	}

}

// Функция, которая складывает поэлементно два массива
void multiplyMatrixes(double* firstArr, double* secondArr, double* finalArr) {

	double row_sum = 0;
	double column_sum = 0;
	double sum = 0;

	int final_array_index = 0;

	// обходим каждую строку первой матрицы
	for (int i = 0; i < arrsize; i++) {
		// обходим каждый столбец второй матрицы
		for (int j = 0; j < arrsize; j++) {
			for (int k = 0; k < arrsize; k++) {
				sum += firstArr[i*k] * secondArr[j*k];
			}

			finalArr[final_array_index] = sum;
			sum = 0;
			final_array_index++;
		}
	}
}

// Функция, которая находит сумму всех элементов в массиве
double sumArrayElems(double* arr)
{
	double sum = 0;

	for (int i = 0; i < arrsize*arrsize; i++)
		sum += arr[i];
	return sum;
}